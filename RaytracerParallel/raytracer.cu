#include "hip/hip_runtime.h"
#include "raytracer.cuh"
#include <cmath>

__device__ __host__
void raytracer::vec(float* a, float* b, float* c)
{
	a[0] = b[0] - c[0];
	a[1] = b[1] - c[1];
	a[2] = b[2] - c[2];
}

__device__ __host__
void raytracer::cross_product(float* a, float* b, float* c)
{
	a[0] = b[1] * c[2] - c[1] * b[2];
	a[1] = b[2] * c[0] - c[2] * b[0];
	a[2] = b[0] * c[1] - c[0] * b[1];
}

__device__ __host__
float raytracer::dot_product(float* a, float* b)
{
	return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}

__device__ __host__
float raytracer::distance_formula_3d(float x1, float y1, float z1, float x2, float y2, float z2)
{
	return sqrt((x1-x2)*(x1-x2)+(y1-y2)*(y1-y2)+(z1-z2)*(z1-z2));
}

__device__ __host__
void raytracer::alpha_beta_to_cartesian(float x1, float y1, float x2, float y2, float percent, float* outx, float* outy)
{
	float dist = sqrt((x1-x2)*(x1-x2)+(y1-y2)*(y1-y2));
	float dist_seg = dist * percent;
	*outx = (x2 - x1) * dist_seg / dist + x1;
	*outy = (y2 - y1) * dist_seg / dist + y1;
}

__device__ __host__
void raytracer::transform_trace_to_uv(triangle* t, vertex* hit)
{	
	//Naming: s->side, i->inner, a|b|c->vertex
	float sa, sb, sc, ia, ib, aa, ab, ad, sad, ida, ie, alpha, beta;
	const float PI = 3.1415926f;
	
	//Calculate distances between triangle sides and hit positions. This flattens the triangle to 2D space.
	sa = distance_formula_3d(t->v[1]->x, t->v[1]->y, t->v[1]->z, t->v[2]->x, t->v[2]->y, t->v[2]->z);
	sb = distance_formula_3d(t->v[0]->x, t->v[0]->y, t->v[0]->z, t->v[2]->x, t->v[2]->y, t->v[2]->z);
	sc = distance_formula_3d(t->v[0]->x, t->v[0]->y, t->v[0]->z, t->v[1]->x, t->v[1]->y, t->v[1]->z);
	ia = distance_formula_3d(t->v[0]->x, t->v[0]->y, t->v[0]->z, hit->x, hit->y, hit->z);
	ib = distance_formula_3d(t->v[1]->x, t->v[1]->y, t->v[1]->z, hit->x, hit->y, hit->z);

	//Calculate alpha and beta.
	aa = acos(((sc*sc)+(ia*ia)-(ib*ib))/(2.0f*sc*ia))/PI*180.0f;
	ab = acos(((sa*sa)+(sc*sc)-(sb*sb))/(2.0f*sa*sc))/PI*180.0f;
	ad = 180.0f-aa-ab;
	sad = sin(ad*PI/180.0f);
	ida = (sc*sin(ab*PI/180.0f))/sad;
	ie = (sc*sin(aa*PI/180.0f))/sad;
	alpha = ia/ida;
	beta = ie/sa;
	
	//Convert alpha and beta to UV coordinates.
	alpha_beta_to_cartesian(t->v[1]->u, t->v[1]->v, t->v[2]->u, t->v[2]->v, beta, &hit->u, &hit->v);
	alpha_beta_to_cartesian(t->v[0]->u, t->v[0]->v, hit->u, hit->v, alpha, &hit->u, &hit->v);
}

//Source of this function only: http://www.lighthouse3d.com/tutorials/maths/ray-triangle-intersection/
__device__ __host__
bool raytracer::process_intersection(float* p, float* d, float* v0, float* v1, float* v2, float* t)
{
	float e1[3], e2[3], h[3], s[3], q[3];
	float a, f, u, v;
	vec(e1,v1,v0);
	vec(e2,v2,v0);
	
	cross_product(h,d,e2);
	a = dot_product(e1,h);
	
	if (a > -0.00001 && a < 0.00001)
	{
		return false;	
	}
	
	f = 1 / a;
	vec(s,p,v0);
	u = f * (dot_product(s,h));
	
	if (u < 0.0 || u > 1.0)
	{
		return false;	
	}
	
	cross_product(q,s,e1);
	v = f * dot_product(d,q);
	
	if (v < 0.0 || u + v > 1.0)
	{
		return false;
	}
	
	//At this stage, we compute t to find out where intersection point is on line.
	*t = f * dot_product(e2,q);
	
	if (*t > 0.00001) //Ray intersection.
	{
		return true;
	}
	else //Line intersection, but not ray intersection.
	{
		return false;
	}
}

__device__ __host__
bool raytracer::get_intersection(vertex* ray, triangle* tri, vertex* intersect, float* t)
{
	float p[3], d[3], v0[3], v1[3], v2[3];
	p[0] = ray->x; p[1] = ray->y; p[2] = ray->z;
	d[0] = ray->nx; d[1] = ray->ny; d[2] = ray->nz;
	v0[0] = tri->v[0]->x; v0[1] = tri->v[0]->y; v0[2] = tri->v[0]->z;
	v1[0] = tri->v[1]->x; v1[1] = tri->v[1]->y; v1[2] = tri->v[1]->z;
	v2[0] = tri->v[2]->x; v2[1] = tri->v[2]->y; v2[2] = tri->v[2]->z;
	
	if (!process_intersection(p, d, v0, v1, v2, t)) //If ray doesn't hit this triangle...
	{
		return false;
	}
	
	(*intersect).x = (p[0] + d[0] * (*t));
	(*intersect).y = (p[1] + d[1] * (*t));
	(*intersect).z = (p[2] + d[2] * (*t));
	return true;
}
